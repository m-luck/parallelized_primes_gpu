// Michael Lukiman at the Courant Institute

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
typedef unsigned int uint;

// GENERATE - PARALLEL VERSION
__global__ void generate( uint N, uint* array ) // Populates an array from 2 to N, assuming an already allocated array space.
{
	uint ind = blockDim.x * blockIdx.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;

	for ( uint nth = ind ; nth < N-1 ; nth += stride )
	{
		array[nth] = 2 + nth;
	}
}
//----------------------------------


// SHOOT-LOOP - PARALLEL VERSION
__global__ void shootLoop( uint N, uint* array, uint numBlocks, uint threadsPerBlock, uint limit ) // Increment through the numbers. If not shot, proceed to shoot using that number. Non-prime numbers will be shot long before they are reached, as can be mathematically induced.
{
	
	uint ind = blockDim.x * blockIdx.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	
	for ( uint loop_index = ind ; loop_index < limit; loop_index+=stride )
	{
		uint current = array[loop_index];
		if ( current != 0 )
		{
			for ( uint nth = (current-2)+current; nth < N-1; nth+=current) 
			{
				array[nth] = 0;
			}
		}
	}
}
//----------------------------------

// MAIN
int main( int argc, char** argv )
{
	uint N = atol(argv[1]);

	if ( argc != 2 || N <= 2 )
	{
		printf("%s", "Please supply one argument, N, for prime numbers up to N. Naturally, N must be greater than 2 for the output to be significant. Thanks! Here's an example: ./genprimes 20\n");
	}

	else
	{
		uint numBlocks = (N + 256 - 2) / 256;
		uint threadsPerBlock = 256;
		uint *device_array;
		uint limit = (N+1) / 2;
		hipMalloc((void **)&device_array, (N-1)*sizeof(uint));
		hipMemset(device_array, 0, (N-1)*sizeof(uint));
		generate<<<numBlocks, threadsPerBlock>>>(N, device_array);
		hipDeviceSynchronize();	
		shootLoop<<<numBlocks,threadsPerBlock>>>(N, device_array, numBlocks, threadsPerBlock, limit);
		hipDeviceSynchronize();
		uint * host_array = new uint[N-1];	
		hipMemcpy(host_array, device_array, sizeof(uint) * (N-1), hipMemcpyDeviceToHost);
		char buffer[1024];
		FILE *outfile;
		snprintf(buffer, sizeof(buffer), "%u.txt", N);
		outfile = fopen(buffer, "a");
		for ( uint i = 0 ; i < N-1 ; i++ )
		{
			if ( host_array[i] != 0 )
			{
				fprintf(outfile, "%u ", host_array[i] );
			}
		//	else { printf( "%u\n", host_array[i] ); }
		}
		hipFree(device_array);
		delete [] host_array;
	}
	return 1;
}

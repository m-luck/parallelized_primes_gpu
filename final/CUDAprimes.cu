// Michael Lukiman at the Courant Institute

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
typedef unsigned int uint;

// GENERATE - PARALLEL VERSION
__global__ void generate( uint N, uint* array ) // Populates an array from 2 to N, assuming an already allocated array space.
{
	uint ind = blockDim.x * blockIdx.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;

	for ( uint nth = ind ; nth < N-1 ; nth += stride )
	{
		array[nth] = 2 + nth;
	}
}
//----------------------------------

// SHOOT - PARALLEL VERSION
__global__ void shoot( uint N, uint multipleOf, uint* array ) // Turns every multipleOf value (except for the number itself) into value 0. Again, index[0] is actually the int 2 and increments from there.
{
	uint ind = blockDim.x * blockIdx.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;

	for ( uint nth = (multipleOf - 2) + (multipleOf*(ind+1)) ; nth < N-1 ; nth += multipleOf*(stride+1) )
	{
		array[nth] = 0;
	}
}
//----------------------------------

// SHOOT-LOOP
void shootLoop( uint N, uint* array, uint numBlocks, uint threadsPerBlock ) // Increment through the numbers. If not shot, proceed to shoot using that number. Non-prime numbers will be shot long before they are reached, as can be mathematically induced.
{
	
	for ( uint loop_index = 0 ; loop_index < floor( ( N - 1 ) / 2 ) ; loop_index++ )
	{
		
		if ( array[loop_index] != 0 )
		{
			shoot<<<numBlocks, threadsPerBlock>>>(N, array[loop_index], array);
			hipDeviceSynchronize();
			
		}
	}
}
//----------------------------------

// MAIN
int main( int argc, char** argv )
{
	uint N = atol(argv[1]);

	if ( argc != 2 || N <= 2 )
	{
		printf("%s", "Please supply one argument, N, for prime numbers up to N. Naturally, N must be greater than 2 for the output to be significant. Thanks! Here's an example: ./genprimes 20\n");
	}

	else
	{
		uint numBlocks = (N + 256 - 2) / 256;
		uint threadsPerBlock = 256;
		uint *device_array;
		hipMallocManaged(&device_array, (N-1)*sizeof(uint));
		hipMemset(device_array, 0, (N-1)*sizeof(uint));
		generate<<<numBlocks, threadsPerBlock>>>(N, device_array);
		hipDeviceSynchronize();	
		shootLoop(N, device_array, numBlocks, threadsPerBlock);
		uint * host_array = new uint[N-1];	
		hipMemcpy(host_array, device_array, sizeof(uint) * (N-1), hipMemcpyDeviceToHost);
		char buffer[1024];
		FILE *outfile;
		snprintf(buffer, sizeof(buffer), "%u.txt", N);
		outfile = fopen(buffer, "a");
		for ( uint i = 0 ; i < N-1 ; i++ )
		{
			if ( host_array[i] != 0 )
			{
				fprintf(outfile, "%u ", host_array[i] );
			}
		//	else { printf( "%u\n", host_array[i] ); }
		}
		hipFree(device_array);
		delete [] host_array;
	}
	return 0;
}

// Michael Lukiman at the Courant Institute

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
typedef unsigned int uint;

// GENERATE - PARALLEL VERSION
__global__ void generate( uint N, uint* array ) // Populates an array from 2 to N, assuming an already allocated array space.
{
	int ind = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for ( int nth = ind ; nth < N-1 ; nth += stride )
	{
		array[nth] = 2 + nth;
	}
}
//----------------------------------

// SHOOT - PARALLEL VERSION
__global__ void shoot( uint N, uint multipleOf, uint* array, int limit ) // Turns every multipleOf value (except for the number itself) into value 0. Again, index[0] is actually the int 2 and increments from there.
{
	int ind = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	while (multipleOf < limit) {
		for ( int nth = (multipleOf-2) + (multipleOf*(ind+1)) ; nth < N-1 ; nth += multipleOf*(stride+1) )
		{	
			array[nth]=0;
		}
		multipleOf++;
	}
}
//----------------------------------

// SHOOT-LOOP
void shootLoop( uint N, uint* array, uint numBlocks, uint threadsPerBlock ) // Increment through the numbers. If not shot, proceed to shoot using that number. Non-prime numbers will be shot long before they are reached, as can be mathematically induced.
{
	int limit = floor( (N+1) / 2); 	
	int multipleGroups = limit/16;
	shoot<<<numBlocks, threadsPerBlock, 0>>>(N, 2, array, multipleGroups);		
	for (int stream = 1; stream < 16; stream++) {
		shoot<<<numBlocks, threadsPerBlock, stream>>>(N, multipleGroups * stream, array, multipleGroups * (stream+1));		
	}

}
//----------------------------------

// MAIN
int main( int argc, char** argv )
{

	uint N = atol(argv[1]);

	if ( argc != 2 || N <= 2 )
	{
		printf("%s", "Please supply one argument, N, for prime numbers up to N. Naturally, N must be greater than 2 for the output to be significant. Thanks! Here's an example: ./genprimes 20\n");
	}

	else
	{
		uint numBlocks = (N + 256 - 2) / 256;
		uint threadsPerBlock = 256;
		uint *device_array;
		hipSetDevice(2);
		hipMalloc((void **) &device_array, (N-1)*sizeof(uint));
		hipMemset(device_array, 0, (N-1)*sizeof(uint));
		generate<<<numBlocks, threadsPerBlock>>>(N, device_array);
		hipDeviceSynchronize();	
		shootLoop(N, device_array, numBlocks, threadsPerBlock);
		uint * host_array = new uint[N-1];	
		hipMemcpy(host_array, device_array, sizeof(uint) * (N-1), hipMemcpyDeviceToHost);
		char buffer[1024];
		FILE *outfile;
		snprintf(buffer, sizeof(buffer), "%u.txt", N);
		outfile = fopen(buffer, "a");
		for ( uint i = 0 ; i < N-1 ; i++ )
		{
			if ( host_array[i] != 0 )
			{
				fprintf(outfile, "%u ", host_array[i] );
			}
		//	else { printf( "%u\n", host_array[i] ); }
		}
		hipFree(device_array);
		delete [] host_array;
	}
	return 0;
}
